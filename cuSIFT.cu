#include "hip/hip_runtime.h"
// TODO: pull out / parameterize magic numbers; maybe cuSIFTOptions?
// TODO: pull out parameters into SiftData so we don't have ridiculously long function signatures
// TODO: compare ScaleDown functions vlfeat
// TODO: add cuImage member variable into SiftData?
// TODO: rename SiftData to cuSIFT
// TODO: double check peak thresh
// TODO: iliagnup 128?

#include "cuSIFT.h"
#include "cuSIFT_D.h"
#include "cuSIFT_D.cu"

SiftData::SiftData(int maxPts, bool host, bool dev) {
  this->numPts = 0;
  this->maxPts = maxPts;

  int numBytes = sizeof(SiftPoint) * maxPts;

#ifdef MANAGEDMEM
  safeCall(hipMallocManaged((void **)&m_data, numBytes));
#else
  h_data = NULL;
  if (host) {
    h_data = (SiftPoint *)malloc(numBytes);
  }

  d_data = NULL;
  if (dev) {
    safeCall(hipMalloc((void **)&d_data, numBytes));
  }
#endif
}

SiftData::~SiftData() {
#ifdef MANAGEDMEM
  safeCall(hipFree(m_data));
#else
  if (d_data != NULL) {
    safeCall(hipFree(d_data));
  }
  d_data = NULL;

  if (h_data != NULL) {
    free(h_data);
  }
  h_data = NULL;
#endif
  numPts = 0;
  maxPts = 0;
}

void SiftData::Synchronize() {
#ifdef MANAGEDMEM
  safeCall(hipDeviceSynchronize());
#else
  if (h_data)
    safeCall(hipMemcpy(h_data, d_data, sizeof(SiftPoint) * numPts, hipMemcpyDeviceToHost));
#endif
}

void SiftData::Extract(float *im, int width, int height, float subsampling) {
  cuImage *cuIm = new cuImage(width, height, im);
  
  TimerGPU timer(0);

  int totPts = 0;

  // Keep track of total number of sift points as well as the maximum number
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_PointCounter), &totPts, sizeof(int)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_MaxNumPoints), &this->maxPts, sizeof(int)));

  // TODO: what is this? why plus 3? This is for temporary space
  // TODO: move NUM_SCALES over
  const int nd = NUM_SCALES + 3;

  // Grab width, height, pitch
  int w = cuIm->width;
  int h = cuIm->height;
  int p = cuIm->pitch;

  int size = 0;             // image sizes
  int sizeTmp = nd * h * p; // laplace buffer sizes

  // Determine how much memory to allocate for extraction
  for (int i = 0; i < numOctaves; i++) {
    w /= 2;
    h /= 2;
    int p = iAlignUp(w, 128);
    size += h * p;
    sizeTmp += nd * h * p;
  }
  float *memoryTmp = NULL;
  size_t pitch;
  size += sizeTmp;

  // TODO: ?? Size = size of height * pitch for each octave + (scales + 3) * height * pitch + ...
  // Return pitch CUDA allocates
  safeCall(hipMallocPitch((void **)&memoryTmp, &pitch, (size_t)4096, (size+4095)/4096*sizeof(float)));

  // TODO: memorySub vs memoryTmp?
  float *memorySub = memoryTmp + sizeTmp;

  ExtractSiftLoop(*cuIm, numOctaves, initBlur, subsampling, memoryTmp, memorySub);
  // ExtractSiftLoop2(cuIm.get(), memoryTmp, memorySub);

  // Copy back number of points found
  safeCall(hipMemcpyFromSymbol(&this->numPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));

  // We should only keep up to maxPts number of SiftPoints
  this->numPts = (this->numPts < this->maxPts ? this->numPts : this->maxPts);
  safeCall(hipFree(memoryTmp));

  Synchronize();
  double totTime = timer.read();
  delete cuIm;

#ifndef VERBOSE
  printf("Total time incl memory =      %.2f ms\n", totTime);
#endif
}

// TODO: bring rootsift back
// void ExtractRootSift(SiftData &siftData, cuImage &img, int numOctaves, double initBlur, float subsampling)
// {
//   TimerGPU timer(0);
//   ExtractSiftHelper(siftData, img, numOctaves, initBlur, thresh, lowestScale, subsampling);
//   ConvertSiftToRootSift(siftData);
//   SynchronizeSift(siftData);
//   double totTime = timer.read();

// #ifndef VERBOSE
//   printf("Total time incl memory =      %.2f ms\n", totTime);
// #endif
// }

// void SiftData::ExtractSiftLoop2(cuImage *img, float *memoryTmp, float *memorySub) {
//   TimerGPU timer(0);

//   float currBlur = initBlur;
//   float currSubsampling = initSubsampling;

//   for (int octaveIndex = 0; octaveIndex < numOctaves; octaveIndex++) {
//     fprintf(stderr, "Processing octave %d\n", octaveIndex);
//     if (lowestScale < currSubsampling * 2.0f) {
//       ExtractSiftOctave(*img, currBlur, peakThresh, lowestScale, currSubsampling, memoryTmp);
//       if (octaveIndex > 0) {
//         delete img;
//       }
//     }

//     int w = img->width / 2;
//     int h = img->height / 2;
//     int p = iAlignUp(w, 128);

//     // TODO: what happens if we have odd w or h?
//     fprintf(stderr, "Making image\n");
//     cuImage *subImg = new cuImage(w, h, memorySub, false);
//     fprintf(stderr, "Scaling down image\n");
//     ScaleDown(*subImg, *img, 0.5f);

//     currBlur = (float)sqrt(currBlur * currBlur + 0.5f * 0.5f) / 2.0f;
//     currSubsampling *= 2.0f;
//     memorySub += h / 2 * p;

//     img = subImg;
//   }

//   double totTime = timer.read();
// #ifdef VERBOSE
//   printf("ExtractSift time total =      %.2f ms\n\n", totTime);
// #endif
// }

// TODO: subsampling? lowest scale?
void SiftData::ExtractSiftLoop(cuImage &img, int numOctaves, double initBlur, float subsampling, float *memoryTmp, float *memorySub) 
{
  TimerGPU timer(0);

  int w = img.width;
  int h = img.height;
  if (numOctaves > 1) {
    cuImage *subImg = new cuImage(w / 2, h / 2, memorySub);
    int p = iAlignUp(w / 2, 128);
    
    ScaleDown(*subImg, img, 0.5f);

    // TODO: Why alls this magicness
    float totInitBlur = (float)sqrt(initBlur * initBlur + 0.5f * 0.5f) / 2.0f;

    ExtractSiftLoop(*subImg, numOctaves - 1, totInitBlur, subsampling * 2.0f, memoryTmp, memorySub + (h / 2) * p);
    delete subImg;
  }

  if (lowestScale<subsampling * 2.0f) {
    ExtractSiftOctave(img, initBlur, subsampling, memoryTmp);
  }

  double totTime = timer.read();
#ifdef VERBOSE
  printf("ExtractSift time total =      %.2f ms\n\n", totTime);
#endif
}

void SiftData::ExtractSiftOctave(cuImage &img, double initBlur, float subsampling, float *memoryTmp)
{
  // TODO: again, what is this?
  const int nd = NUM_SCALES + 3;
  TimerGPU timer0;
  cuImage diffImg[nd];
  int w = img.width;
  int h = img.height;
  int p = img.pitch;
  for (int i = 0; i < nd - 1; i++) {
    diffImg[i].Allocate(w, h, p, false, memoryTmp + i * p * h);
  }

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = img.d_data;
  resDesc.res.pitch2D.width = img.width;
  resDesc.res.pitch2D.height = img.height;
  resDesc.res.pitch2D.pitchInBytes = img.pitch*sizeof(float);  
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0]   = hipAddressModeClamp;
  texDesc.addressMode[1]   = hipAddressModeClamp;
  texDesc.filterMode       = hipFilterModeLinear;
  texDesc.readMode         = hipReadModeElementType;
  texDesc.normalizedCoords = 0;
  // Create texture object
  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  TimerGPU timer1;
  float baseBlur = pow(2.0f, -1.0f/NUM_SCALES);
  float diffScale = pow(2.0f, 1.0f/NUM_SCALES);
  LaplaceMulti(texObj, diffImg, baseBlur, diffScale, initBlur);
  int fstPts = 0;
  safeCall(hipMemcpyFromSymbol(&fstPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  double sigma = baseBlur*diffScale;

  // Pull out thresholds
  FindPointsMulti(diffImg, sigma, 1.0f/NUM_SCALES, subsampling);
  double gpuTimeDoG = timer1.read();
  TimerGPU timer4;
  int totPts = 0;
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts < this->maxPts ? totPts : this->maxPts);
  if (totPts>fstPts) {
    ComputeOrientations(texObj, fstPts, totPts); 
    safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
    totPts = (totPts < this->maxPts ? totPts : this->maxPts);
    ExtractSiftDescriptors(texObj, fstPts, totPts, subsampling); 
  }
  safeCall(hipDestroyTextureObject(texObj));
  double gpuTimeSift = timer4.read();

  double totTime = timer0.read();
#ifdef VERBOSE
  printf("GPU time : %.2f ms + %.2f ms + %.2f ms = %.2f ms\n", totTime-gpuTimeDoG-gpuTimeSift, gpuTimeDoG, gpuTimeSift, totTime);
  safeCall(hipMemcpyFromSymbol(&totPts, HIP_SYMBOL(d_PointCounter), sizeof(int)));
  totPts = (totPts < this->maxPts ? totPts : this->maxPts);
  if (totPts>0) 
    printf("           %.2f ms / DoG,  %.4f ms / Sift,  #Sift = %d\n", gpuTimeDoG/NUM_SCALES, gpuTimeSift/(totPts-fstPts), totPts-fstPts); 
#endif
}

// void InitSiftData(SiftData &data, int num, bool host, bool dev)
// {
//   data.numPts = 0;
//   data.maxPts = num;
//   int sz = sizeof(SiftPoint)*num;
// #ifdef MANAGEDMEM
//   safeCall(hipMallocManaged((void **)&data.m_data, sz));
// #else
//   data.h_data = NULL;
//   if (host)
//     data.h_data = (SiftPoint *)malloc(sz);
//   data.d_data = NULL;
//   if (dev)
//     safeCall(hipMalloc((void **)&data.d_data, sz));
// #endif
// }

// void FreeSiftData(SiftData &data)
// {
// #ifdef MANAGEDMEM
//   safeCall(hipFree(data.m_data));
// #else
//   if (data.d_data!=NULL)
//     safeCall(hipFree(data.d_data));
//   data.d_data = NULL;
//   if (data.h_data!=NULL)
//     free(data.h_data);
//   data.h_data = NULL;
// #endif
//   data.numPts = 0;
//   data.maxPts = 0;
// }

///////////////////////////////////////////////////////////////////////////////
// Host side master functions
///////////////////////////////////////////////////////////////////////////////

// General strategy outlined here: http://docs.nvidia.com/cuda/samples/3_Imagi
// ng/convolutionSeparable/doc/convolutionSeparable.pdf
// TODO: convert to cuImage member function?
// TODO: investigate SCALEDOWN_W warps (160 x 16) -> chosen for apron size
double ScaleDown(cuImage &res, cuImage &src, float variance) {
  // Make sure we have allocated device data for both source and resource
  if (res.d_data == NULL || src.d_data == NULL) {
    printf("ScaleDown: missing data\n");
    return 0.0;
  }

  // 5-pixel linear gaussian kernel
  float h_Kernel[5];

  // Normalizing factor
  float kernelSum = 0.0f;

  // Compute kernel values. We only access n / 2 + 1 of the values in the CUDA
  // kernel, but we compute all of the results so that we can normalize. We
  // can do without computing the extra exp here as well, but maybe this was
  // just easier?
  for (int j = 0; j < 5; j++) {
    h_Kernel[j] = (float)expf(-(double)(j - 2) * (j - 2) / 2.0 / variance);
    kernelSum += h_Kernel[j];
  }

  // Normalize kernel values by kernelSum
  for (int j = 0; j < 5; j++) {
    h_Kernel[j] /= kernelSum;  
  }

  // Pass kernel to device
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel1), h_Kernel, 5 * sizeof(float)));

  // Set number of blocks and threads; chosen for apron size (i.e., because we
  // are doing convolution, we need to access memory outside a block; to make
  // this efficient, we want to bring in data outside the block)
  dim3 blocks(iDivUp(src.width, SCALEDOWN_W), iDivUp(src.height, SCALEDOWN_H));
  dim3 threads(SCALEDOWN_W + 4);

  // Start CUDA kernel
  ScaleDown_D<<<blocks, threads>>>(res.d_data, src.d_data, src.width, src.pitch, src.height, res.pitch); 
  checkMsg("ScaleDown() execution failed\n");
  return 0.0;
}

double SiftData::ComputeOrientations(hipTextureObject_t texObj, int fstPts, int totPts) {
  dim3 blocks(totPts - fstPts);
  dim3 threads(128);
#ifdef MANAGEDMEM
  ComputeOrientations_D<<<blocks, threads>>>(texObj, m_data, fstPts);
#else
  ComputeOrientations_D<<<blocks, threads>>>(texObj, d_data, fstPts);
#endif
  checkMsg("ComputeOrientations_D() execution failed\n");
  return 0.0;
}

double SiftData::ExtractSiftDescriptors(hipTextureObject_t texObj, int fstPts, int totPts, float subsampling) {
  dim3 blocks(totPts - fstPts); 
  dim3 threads(16, 8);
#ifdef MANAGEDMEM
  ExtractSiftDescriptors_D<<<blocks, threads>>>(texObj, m_data, fstPts, subsampling);
#else
  ExtractSiftDescriptors_D<<<blocks, threads>>>(texObj, d_data, fstPts, subsampling);
#endif
  checkMsg("ExtractSiftDescriptors_D() execution failed\n");
  return 0.0; 
}


// TODO: Really, we should reimplement the end of ExtractSiftDescriptors in
// cudaSiftD.cu so we don't do L2 normalization and then L1 normalization in
// the case of RootSift
double SiftData::ConvertSiftToRootSift() {
  // For now, do naive parallelization. We are essentially creating a for loop
  // over all the sift points
  dim3 blocks(iDivUp(numPts, 16));
  dim3 threads(16);
#ifdef MANAGEDMEM
  ConvertSiftToRootSift_D<<<blocks, threads>>>(m_data, numPts);
#else
  ConvertSiftToRootSift_D<<<blocks, threads>>>(d_data, numPts);
#endif
  checkMsg("ConvertSiftToRootSift_D() execution failed\n");
  return 0.0; 
}

//==================== Multi-scale functions ===================//

double SiftData::LaplaceMulti(hipTextureObject_t texObj, cuImage *results, float baseBlur, float diffScale, float initBlur) {
  float kernel[12*16];
  float scale = baseBlur;
  for (int i=0;i<NUM_SCALES+3;i++) {
    float kernelSum = 0.0f;
    float var = scale*scale - initBlur*initBlur;
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) {
      kernel[16*i+j+LAPLACE_R] = (float)expf(-(double)j*j/2.0/var);
      kernelSum += kernel[16*i+j+LAPLACE_R]; 
    }
    for (int j=-LAPLACE_R;j<=LAPLACE_R;j++) 
      kernel[16*i+j+LAPLACE_R] /= kernelSum;  
    scale *= diffScale;
  }
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel2), kernel, 12*16*sizeof(float)));
  int width = results[0].width;
  int pitch = results[0].pitch;
  int height = results[0].height;
  dim3 blocks(iDivUp(width+2*LAPLACE_R, LAPLACE_W), height);
  dim3 threads(LAPLACE_W+2*LAPLACE_R, LAPLACE_S); 
  LaplaceMulti_D<<<blocks, threads>>>(texObj, results[0].d_data, width, pitch, height);
  checkMsg("LaplaceMulti_D() execution failed\n");
  return 0.0; 
}

double SiftData::FindPointsMulti(cuImage *sources, float scale, float factor, float subsampling) {
  if (sources->d_data==NULL) {
    printf("FindPointsMulti: missing data\n");
    return 0.0;
  }
  int w = sources->width;
  int p = sources->pitch;
  int h = sources->height;
  float threshs[2] = { peakThresh, -peakThresh };
  float scales[NUM_SCALES];  
  float diffScale = pow(2.0f, factor);
  for (int i=0;i<NUM_SCALES;i++) {
    scales[i] = scale;
    scale *= diffScale;
  }

  // TODO: rename d_Threshold, d_EdgeLimit?
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Threshold), &threshs, 2*sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeLimit), &edgeThresh, sizeof(float)));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Scales), scales, sizeof(float)*NUM_SCALES));
  safeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_Factor), &factor, sizeof(float)));

  dim3 blocks(iDivUp(w, MINMAX_W)*NUM_SCALES, iDivUp(h, MINMAX_H));
  dim3 threads(MINMAX_W + 2); 
#ifdef MANAGEDMEM
  FindPointsMulti_D<<<blocks, threads>>>(sources->d_data, m_data, w, p, h, NUM_SCALES, subsampling); 
#else
  FindPointsMulti_D<<<blocks, threads>>>(sources->d_data, d_data, w, p, h, NUM_SCALES, subsampling); 
#endif
  checkMsg("FindPointsMulti_D() execution failed\n");
  return 0.0;
}

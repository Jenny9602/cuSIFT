#include "hip/hip_runtime.h"
#include "cudaSift.h"
#include "cudautils.h"

//================= Device matching functions =====================//

__global__ void MatchSiftPoints(SiftPoint *sift1, SiftPoint *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoint[128];
  __shared__ float sums[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x;
  const int p2 = blockIdx.y*16 + ty;
  const float *ptr1 = sift1[p1].data;
  const float *ptr2 = sift2[p2].data;
  const int i = 16*ty + tx;
  if (ty<8)
    siftPoint[i] = ptr1[i];
  __syncthreads();
  float sum = 0.0f;
  if (p2<numPts2)
    for (int j=0;j<8;j++)
      sum += siftPoint[16*j+tx] * ptr2[16*j+tx];
  sums[i] = sum;
  __syncthreads();
  if (tx<8)
    sums[i] += sums[i+8];
  __syncthreads();
  if (tx<4)
    sums[i] += sums[i+4];
  __syncthreads();
  if (ty==0) {
    sum = sums[16*tx+0] + sums[16*tx+1] + sums[16*tx+2] + sums[16*tx+3];
    corrData[p1*gridDim.y*16 + blockIdx.y*16 + tx] = sum;
  }
  __syncthreads();
}

__global__ void MatchSiftPoints2(SiftPoint *sift1, SiftPoint *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoints1[16*128];
  __shared__ float siftPoints2[16*128];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const float *ptr1 = sift1[min(numPts1-1,blockIdx.x*16 + ty)].data;
  const float *ptr2 = sift2[min(numPts2-1,blockIdx.y*16 + ty)].data;
  for (int i=0;i<8;i++) {
    siftPoints1[128*ty+16*i+tx] = ptr1[16*i+tx];
    siftPoints2[128*ty+16*i+tx] = ptr2[16*i+tx];
  }
  __syncthreads();
  const int p1 = blockIdx.x*16 + ty;
  const int p2 = blockIdx.y*16 + tx;
  const float *pt1 = &siftPoints1[ty*128];
  const float *pt2 = &siftPoints2[tx*128];
  float sum = 0.0f;
  for (int i=0;i<128;i++) {
    int itx = (i + tx)&127; // avoid bank conflicts
    sum += pt1[itx]*pt2[itx];
  }
  if (p1<numPts1)
    corrData[p1*gridDim.y*16 + p2] = (p2<numPts2 ? sum : -1.0f);
}

__global__ void FindMaxCorr(float *corrData, SiftPoint *sift1, SiftPoint *sift2, int numPts1, int corrWidth, int siftSize)
{
  __shared__ float maxScore[16*16];
  __shared__ float maxScor2[16*16];
  __shared__ int maxIndex[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = ty*16 + tx;
  int p1 = blockIdx.x*16 + threadIdx.y;
  p1 = (p1>=numPts1 ? numPts1-1 : p1);
  maxScore[idx] = -1.0f;
  maxScor2[idx] = -1.0f;
  maxIndex[idx] = -1;
  __syncthreads();
  float *corrs = &corrData[p1*corrWidth];
  for (int i=tx;i<corrWidth;i+=16) {
    float val = corrs[i];
    if (val>maxScore[idx]) {
      maxScor2[idx] = maxScore[idx];
      maxScore[idx] = val;
      maxIndex[idx] = i;
    } else if (val>maxScor2[idx])
      maxScor2[idx] = val;
  }
  //if (p1==1)
  //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
  //	   tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  __syncthreads();
  for (int len=8;len>0;len/=2) {
    if (tx<8) {
      float val = maxScore[idx+len];
      int i = maxIndex[idx+len];
      if (val>maxScore[idx]) {
	maxScor2[idx] = maxScore[idx];
	maxScore[idx] = val;
	maxIndex[idx] = i;
      } else if (val>maxScor2[idx])
	maxScor2[idx] = val;
      float va2 = maxScor2[idx+len];
      if (va2>maxScor2[idx])
	maxScor2[idx] = va2;
    }
    __syncthreads();
    //if (p1==1 && tx<len) 
    //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
    //	     tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  }
  if (tx==6)
    sift1[p1].score = maxScore[ty*16];
  if (tx==7)
    sift1[p1].ambiguity = maxScor2[ty*16] / (maxScore[ty*16] + 1e-6);
  if (tx==8)
    sift1[p1].match = maxIndex[ty*16];
  if (tx==9)
    sift1[p1].match_xpos = sift2[maxIndex[ty*16]].xpos;
  if (tx==10)
    sift1[p1].match_ypos = sift2[maxIndex[ty*16]].ypos;
  __syncthreads();
  //if (tx==0)
  //  printf("index = %d/%d, score = %.2f, ambiguity = %.2f, match = %d\n", 
  //	p1, numPts1, sift1[p1].score, sift1[p1].ambiguity, sift1[p1].match);
}

double MatchSiftData(SiftData &data1, SiftData &data2)
{
  TimerGPU timer(0);
  int numPts1 = data1.numPts;
  int numPts2 = data2.numPts;
  if (!numPts1 || !numPts2) 
    return 0.0;
#ifdef MANAGEDMEM
  SiftPoint *sift1 = data1.m_data;
  SiftPoint *sift2 = data2.m_data;
#else
  if (data1.d_data==NULL || data2.d_data==NULL)
    return 0.0f;
  SiftPoint *sift1 = data1.d_data;
  SiftPoint *sift2 = data2.d_data;
#endif
  
  float *d_corrData; 
  int corrWidth = iDivUp(numPts2, 16)*16;
  int corrSize = sizeof(float)*numPts1*corrWidth;
  safeCall(hipMalloc((void **)&d_corrData, corrSize));
#if 0
  dim3 blocks1(numPts1, iDivUp(numPts2, 16));
  dim3 threads1(16, 16); // each block: 1 points x 16 points
  MatchSiftPoints<<<blocks1, threads1>>>(sift1, sift2, d_corrData, numPts1, numPts2);
#else
  dim3 blocks(iDivUp(numPts1,16), iDivUp(numPts2, 16));
  dim3 threads(16, 16); // each block: 1 points x 16 points
  MatchSiftPoints2<<<blocks, threads>>>(sift1, sift2, d_corrData, numPts1, numPts2);
#endif
  safeCall(hipDeviceSynchronize());
  dim3 blocksMax(iDivUp(numPts1, 16));
  dim3 threadsMax(16, 16);
  FindMaxCorr<<<blocksMax, threadsMax>>>(d_corrData, sift1, sift2, numPts1, corrWidth, sizeof(SiftPoint));
  safeCall(hipDeviceSynchronize());
  checkMsg("MatchSiftPoints() execution failed\n");
  safeCall(hipFree(d_corrData));
  if (data1.h_data!=NULL) {
    float *h_ptr = &data1.h_data[0].score;
    float *d_ptr = &data1.d_data[0].score;
    safeCall(hipMemcpy2D(h_ptr, sizeof(SiftPoint), d_ptr, sizeof(SiftPoint), 5*sizeof(float), data1.numPts, hipMemcpyDeviceToHost));
  }

  double gpuTime = timer.read();
#ifdef VERBOSE
  printf("MatchSiftData time =          %.2f ms\n", gpuTime);
#endif
  return gpuTime;
}
